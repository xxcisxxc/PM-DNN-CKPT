#include "persist_param.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void cudaInfo(int *optimal_size, int *lprior, int *hprior, Py_ssize_t nbytes)
{
    const int max_streams = 100;
    hipDeviceGetStreamPriorityRange(lprior, hprior);

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    *optimal_size = devProp.maxThreadsPerBlock;
    if (nbytes > (*optimal_size * max_streams))
    {
        *optimal_size = (int)(nbytes / (8 * max_streams)) * 8;
    }
}

struct StreamInfo
{
    hipStream_t stream;
    Py_ssize_t offset;
    Py_ssize_t length;
};

struct CoWStreamInfo
{
    hipStream_t stream;
    char *part_tensor;
};

void PersistParam::_set_fromall(std::string name, char *oldtensor, Py_ssize_t nbytes, Py_ssize_t *flag)
{
    BaseAddrInfo &BAInfo = baseaddr.at(name);
    const char *newtensor = BAInfo.addr;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostRegister(oldtensor, nbytes, hipHostRegisterMapped);

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    hipMemcpyAsync(oldtensor, newtensor, nbytes, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    BAInfo.cur_pos = 2;

    hipHostUnregister(oldtensor);
    //hipMemcpy(oldtensor, newtensor, nbytes, hipMemcpyDeviceToHost);

    pmemobj_persist(pop, oldtensor, nbytes);

    *flag = 1;
    pmemobj_persist(pop, flag, PY_SSIZE_BYTES);
}

void PersistParam::_set_frombase(std::string name, char *oldtensor, Py_ssize_t nbytes, Py_ssize_t *flag)
{
    int optimal_size, lprior, hprior;
    cudaInfo(&optimal_size, &lprior, &hprior, nbytes);
    
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostRegister(oldtensor, nbytes, hipHostRegisterMapped);

    Py_ssize_t num_streams = NBytesM(nbytes, optimal_size);
    std::vector<StreamInfo> copy_streams(num_streams);
    
    BaseAddrInfo &BAInfo = baseaddr.at(name);
    BAInfo.cur_pos = 0;
    char *addr = BAInfo.addr;
    Py_ssize_t cur_pos = nbytes % optimal_size;
    if (cur_pos == 0) cur_pos = optimal_size;

    hipStreamCreateWithFlags(&(copy_streams[0].stream), hipStreamNonBlocking);
    hipMemcpyAsync(oldtensor, addr, cur_pos, hipMemcpyDeviceToHost, copy_streams[0].stream);
    copy_streams[0].offset = 0;
    copy_streams[0].length = cur_pos;
    for (Py_ssize_t i = 1; i < num_streams; cur_pos += optimal_size, i++)
    {
        hipStreamCreateWithFlags(&(copy_streams[0].stream), hipStreamNonBlocking);
        hipMemcpyAsync(oldtensor+cur_pos, addr+cur_pos, optimal_size, hipMemcpyDeviceToHost, copy_streams[i].stream);
        copy_streams[i].offset = cur_pos;
        copy_streams[i].length = optimal_size;
    }

    std::vector<CoWStreamInfo> CoW_streams;
    std::vector<StreamInfo> Remain_streams;
    while (!copy_streams.empty())
    {
        Py_ssize_t i = 0;
        while (i < copy_streams.size())
        {
            if (hipStreamQuery(copy_streams[i].stream) == hipSuccess)
            {
                hipStreamDestroy(copy_streams[i].stream);
                copy_streams.erase(copy_streams.begin() + i);
                continue;
            }
            else
            {
                if (BAInfo.cur_pos == 1)
                {
                    char *part_tensor;
                    hipMalloc(&part_tensor, copy_streams[i].length);

                    hipStream_t highprior_stream;
                    hipStreamCreateWithPriority(&highprior_stream, hipStreamNonBlocking, hprior);
                    hipMemcpyAsync(part_tensor, addr+copy_streams[i].offset, copy_streams[i].length, hipMemcpyDeviceToDevice, highprior_stream);

                    CoW_streams.push_back({highprior_stream, part_tensor});
                    hipMemcpyAsync(oldtensor+copy_streams[i].offset, part_tensor, copy_streams[i].length, hipMemcpyDeviceToHost, copy_streams[i].stream);

                    Remain_streams.push_back(copy_streams[i]);
                    copy_streams.erase(copy_streams.begin() + i);
                }
                else
                    i += 1;
            }
        }
    }

    for (CoWStreamInfo &CoWS : CoW_streams)
    {
        hipStreamSynchronize(CoWS.stream);
        hipStreamDestroy(CoWS.stream);
    }
    BAInfo.cur_pos = 2;
    
    for (Py_ssize_t i = 0; i < Remain_streams.size(); i++)
    {
        hipStreamSynchronize(Remain_streams[i].stream);
        hipStreamDestroy(Remain_streams[i].stream);
        hipFree(CoW_streams[i].part_tensor);
    }

    hipHostUnregister(oldtensor);

    pmemobj_persist(pop, oldtensor, nbytes);

    *flag = 1;
    pmemobj_persist(pop, flag, PY_SSIZE_BYTES);
}

void PersistParam::change_base(py::str name)
{
    BaseAddrInfo &BAInfo = baseaddr.at(name);

    if (BAInfo.cur_pos == 2)
    {
        return;
    }
    BAInfo.cur_pos = 1;
    while (BAInfo.cur_pos != 2);
}